
#include <hip/hip_runtime.h>
/*
	all kernel functions required to run CLAHE3D through CUDA
	Author: Urs Hofmann
	Mail: mail@hofmannu.org
	Date: 13.02.2022
*/

// const arguments passed to equilization kernel 
#ifndef EQ_ARGUMENTS_H
#define EQ_ARGUMENTS_H

struct eq_arguments
{
	uint64_t volSize[3]; // total size of volume
	float origin[3]; // origin of the subvolume grid
	float end[3]; // end of the subvolume grid
	uint64_t nSubVols[3]; // number of subvolumes
	uint64_t spacingSubVols[3]; // spacing between subvolumes
	float* minValBin; // minimum value in each bib
	float* maxValBin; // maximum value in each bin
	float* cdf; // cummulative distribution function
	uint64_t nBins; // number of bins we have for the histogram
};

#endif

// returns the indices of the neighbouring subvolumes for a defined position
__device__ inline void get_neighbours_gpu(
	const uint64_t* position,
	uint64_t* neighbours,
	float* ratio,
	const eq_arguments inArgs
	)
{
	#pragma unroll
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		// let see if we hit the lower limit
		if (((float) position[iDim]) <=  inArgs.origin[iDim])
		{
			ratio[iDim] = 0;
			neighbours[iDim * 2] = 0; // left index along current dimension
			neighbours[iDim * 2 + 1] = 0; // right index along current dimension
		}
		else if (((float) position[iDim]) >=  inArgs.end[iDim])
		{
			ratio[iDim] = 0;
			neighbours[iDim * 2] =  inArgs.nSubVols[iDim] - 1; // left index along curr dimension
		 	neighbours[iDim * 2 + 1] =   inArgs.nSubVols[iDim] - 1; // right index along curr dimension
		} 
		else // we are actually in between!
		{
			const float offsetDistance = (float) position[iDim] - (float) inArgs.origin[iDim];
			neighbours[iDim * 2] = (uint64_t) (offsetDistance / inArgs.spacingSubVols[iDim]);
			neighbours[iDim * 2 + 1] = neighbours[iDim * 2] + 1;
			const float leftDistance = offsetDistance - ((float) neighbours[iDim * 2]) * 
				((float) inArgs.spacingSubVols[iDim]);
			ratio[iDim] = leftDistance / ((float) inArgs.spacingSubVols[iDim]);
		}

	}
	return;
}

// return bin in which current value is positioned
__device__ inline float get_icdf_gpu(
	const uint64_t iZ, // index of subvolume we request along z
	const uint64_t iX, // index of subvolume we request along x
	const uint64_t iY, // index of subvolume we request along y
	const float currValue,
	const eq_arguments& inArgs)
{
	// if we are below noise level, directy return 0
	const uint64_t subVolIdx = iZ + inArgs.nSubVols[0] * (iX + inArgs.nSubVols[1] * iY);
	if (currValue <= inArgs.minValBin[subVolIdx])
	{
		return 0.0;
	}
	else
	{
		// get index describes the 3d index of the subvolume
		const uint64_t subVolOffset = inArgs.nBins * subVolIdx;
		const float vInterp = (currValue - inArgs.minValBin[subVolIdx]) / 
			(inArgs.maxValBin[subVolIdx] - inArgs.minValBin[subVolIdx]);
		
		// it can happen that the voxel value is higher then the max value detected
		// in the neighbouring histogram. In this case we crop it to the maximum permittable value
		const uint64_t binOffset = (vInterp > 1.0) ? 
			(inArgs.nBins - 1 + subVolOffset)
			: fmaf(vInterp, (float) inArgs.nBins - 1.0, 0.5) + subVolOffset;

		return inArgs.cdf[binOffset];
	}
}

// kernel function to run equilization
__global__ void equalize_kernel(
	float* dataMatrix, // input and output volume
	const eq_arguments inArgs // constant arguemtns
	)
{
	// get index of currently adjusted voxel
	const uint64_t idxVol[3] = {
		threadIdx.x + blockIdx.x * blockDim.x,
		threadIdx.y + blockIdx.y * blockDim.y,
		threadIdx.z + blockIdx.z * blockDim.z
	};

	if ( // check if we are within boundaries
		(idxVol[0] < inArgs.volSize[0]) && 
		(idxVol[1] < inArgs.volSize[1]) && 
		(idxVol[2] < inArgs.volSize[2]))
	{
		const uint64_t idxVolLin = idxVol[0] + inArgs.volSize[0] * 
			(idxVol[1] + inArgs.volSize[1] * idxVol[2]);
		const float currValue = dataMatrix[idxVolLin];

		// get neighbours defined as the subvolume indices at lower and upper end
		uint64_t neighbours[6];
		float ratio[3];
		get_neighbours_gpu(idxVol, neighbours, ratio, inArgs);
		
		// get values from all eight corners
		const float value[8] = {
			get_icdf_gpu(neighbours[0], neighbours[2], neighbours[4], currValue, inArgs),
			get_icdf_gpu(neighbours[0], neighbours[2], neighbours[5], currValue, inArgs),
			get_icdf_gpu(neighbours[0], neighbours[3], neighbours[4], currValue, inArgs),
			get_icdf_gpu(neighbours[0], neighbours[3], neighbours[5], currValue, inArgs),
			get_icdf_gpu(neighbours[1], neighbours[2], neighbours[4], currValue, inArgs),
			get_icdf_gpu(neighbours[1], neighbours[2], neighbours[5], currValue, inArgs),
			get_icdf_gpu(neighbours[1], neighbours[3], neighbours[4], currValue, inArgs),
			get_icdf_gpu(neighbours[1], neighbours[3], neighbours[5], currValue, inArgs)};
		
		// trilinear interpolation
		dataMatrix[idxVolLin] =
			fmaf(1 - ratio[0], 
				fmaf(1 - ratio[1], 
					fmaf(value[0], 1 - ratio[2], value[1] * ratio[2])
					, ratio[1] * fmaf(value[2], 1 - ratio[2], value[3] * ratio[2])
			), 
			ratio[0] * 
				fmaf(1 - ratio[1],
					fmaf(value[4], 1 - ratio[2], value[5] * ratio[2])
				, ratio[1] * fmaf(value[6], 1 - ratio[2], value[7] * ratio[2])
			));
		}
	return;
}

// struct holding arguments used in cdf kernel
#ifndef CDF_ARGUMENTS_H
#define CDF_ARGUMENTS_H

struct cdf_arguments
{
	uint64_t spacingSubVols[3]; // distance between subvolumes [z, x, y]
	uint64_t nSubVols[3]; // number of subvolumes [z, x, y]
	uint64_t volSize[3]; // overall size of data volume [z, x, y]
	int64_t range[3]; // range of each bin in each direction [z, x, y]
	uint64_t nBins; // number of bins which we use for our histogram
	float noiseLevel; // noise level in matrix
	float origin[3];
};

#endif

// get start index limited by 0
__device__ inline uint64_t get_startIndex(const uint64_t zCenter, const int zRange)
{
	const uint64_t startIdx = (((int) zCenter - zRange) < 0) ? 0 : zCenter - zRange;
	return startIdx;
}

// get stop index limited by volume size
__device__ inline uint64_t get_stopIndex(const uint64_t zCenter, const int zRange, const uint64_t volSize)
{
	const uint64_t stopIdx = (((int) zCenter + zRange) >= volSize) ? volSize : zCenter + zRange;
	return stopIdx;
}

// return cummulative distribution function
__global__ void cdf_kernel(
		float* cdf, 
		float* maxValBin, 
		float* minValBin, 
		const float* dataMatrix,
		const cdf_arguments inArgs
	)
{
	const uint64_t iSub[3] = {
		threadIdx.x + blockIdx.x * blockDim.x,
		threadIdx.y + blockIdx.y * blockDim.y,
		threadIdx.z + blockIdx.z * blockDim.z
	};

	if (
		(iSub[0] < inArgs.nSubVols[0]) && 
		(iSub[2] < inArgs.nSubVols[2]) && 
		(iSub[1] < inArgs.nSubVols[1]))
	{
		// get start and stop indices for currently used bin
		uint64_t startIdx[3];
		uint64_t stopIdx[3];
		#pragma unroll
		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			const float ctr = ((float) iSub[iDim]) * ((float) inArgs.spacingSubVols[iDim]) + inArgs.origin[iDim];
			startIdx[iDim] = get_startIndex(ctr, inArgs.range[iDim]);
			stopIdx[iDim] = get_stopIndex(ctr, inArgs.range[iDim], inArgs.volSize[iDim]);
		}
		
		// index of currently employed subvolume
		const uint64_t idxSubVol = iSub[0] + inArgs.nSubVols[0] * (iSub[1] + inArgs.nSubVols[1] * iSub[2]);
		float* localCdf = &cdf[inArgs.nBins * idxSubVol]; // histogram of subvolume, only temporarily requried
		// volume is indexed as iz + ix * nz + iy * nx * nz
		// cdf is indexed as [iBin, iZSub, iXSub, iYSub]

		// reset bins to zero before summing them up
		for (uint64_t iBin = 0; iBin < inArgs.nBins; iBin++)
			localCdf[iBin] = 0;

		// calculate local maximum and minimum
		const float firstVal = dataMatrix[
			startIdx[0] + inArgs.volSize[0] * (startIdx[1] + inArgs.volSize[1] * startIdx[2])];
		float tempMax = firstVal; // temporary variable to reduce data access
		float tempMin = firstVal;
		for (uint64_t iY = startIdx[2]; iY <= stopIdx[2]; iY++)
		{
			const uint64_t yOffset = iY * inArgs.volSize[0] * inArgs.volSize[1];
			for(uint64_t iX = startIdx[1]; iX <= stopIdx[1]; iX++)
			{
				const uint64_t xOffset = iX * inArgs.volSize[0];
				for(uint64_t iZ = startIdx[0]; iZ <= stopIdx[0]; iZ++)
				{
					const float currVal = dataMatrix[iZ + xOffset + yOffset];
					
					if (currVal > tempMax)
					{
						tempMax = currVal;
					}
					if (currVal < tempMin)
					{
						tempMin = currVal;
					}
				}
			}
		}

		tempMax = (tempMax < inArgs.noiseLevel) ? inArgs.noiseLevel : tempMax;
		maxValBin[idxSubVol] = tempMax;

		tempMin = (tempMin < inArgs.noiseLevel) ? inArgs.noiseLevel : tempMin;
		minValBin[idxSubVol] = tempMin;

		// calculate size of each bin
		const float binRange = (tempMin == tempMax) ? 
			1 : (tempMax - tempMin) / ((float) inArgs.nBins);

		// sort values into bins which are above clipLimit
		for (uint64_t iY = startIdx[2]; iY <= stopIdx[2]; iY++)
		{
			const uint64_t yOffset = iY * inArgs.volSize[0] * inArgs.volSize[1];
			for(uint64_t iX = startIdx[1]; iX <= stopIdx[1]; iX++)
			{
				const uint64_t xOffset = iX * inArgs.volSize[0];
				for(uint64_t iZ = startIdx[0]; iZ <= stopIdx[0]; iZ++)
				{
					const float currVal = dataMatrix[iZ + xOffset + yOffset]; 
					// only add to histogram if above clip limit
					if (currVal >= inArgs.noiseLevel)
					{
						uint64_t iBin = (currVal - tempMin) / binRange;

						// special case for maximum values in subvolume (they gonna end up
						// one index above)
						if (iBin >= inArgs.nBins)
						{
							localCdf[inArgs.nBins - 1] += 1;
						}
						else
						{
							localCdf[iBin] += 1;
						}
					}
				}
			}
		}

		// calculate cummulative sum and scale along y
		float cdfTemp = 0;
		const float zeroElem = localCdf[0];
		for (uint64_t iBin = 0; iBin < inArgs.nBins; iBin++)
		{
			cdfTemp += localCdf[iBin];
			localCdf[iBin] = cdfTemp - zeroElem;
		}

		// now we scale cdf to max == 1 (first value is 0 anyway)
		const float cdfMax = localCdf[inArgs.nBins - 1];
		if (cdfMax > 0)
		{
			for (uint64_t iBin = 1; iBin < inArgs.nBins; iBin++)
			{
				localCdf[iBin] /= cdfMax;
			}
		}
		else
		{
			for (uint64_t iBin = 1; iBin < inArgs.nBins; iBin++)
			{
				localCdf[iBin] = ((float) iBin) / ((float) inArgs.nBins);
			}
		}
	}
	return;
}


