#include "cudaTools.cuh"

cudaTools::cudaTools()
{
	// get number of connected devices
	hipError_t err = hipGetDeviceCount(&nDevices);
	if (err != hipSuccess)
	{
		printf("Could not receive a device count\n");
		throw "CudaError";
	}

	// query device properties for each attached GPU
	hipDeviceProp_t currProps;
	for (int iDevice = 0; iDevice < nDevices; iDevice++)
	{
		err = hipGetDeviceProperties(&currProps, iDevice);
		if (err != hipSuccess)
		{
			printf("Something went wrong while returning properties of device %d\n", iDevice);
			throw "CudaError";
		}

		props.push_back(std::move(currProps));
	}
	return;
}

void cudaTools::print_devProps()
{
	for (int iDevice = 0; iDevice < nDevices; iDevice++)
		print_devProps(iDevice);
	return;
}

// prints the properties of a device to the terminal
void cudaTools::print_devProps(const int iDevice)
{
	if (iDevice < 0)
	{
		printf("Device index must be larger or equal 0!\n");
		throw "InvalidValue";
	}

	if (iDevice >= nDevices)
	{
		printf("Requested ID is exceedin available device number\n");
		throw "InvalidValue";
	}

	printf("General information for device %d: \n", iDevice);
	printf(" - Name: %s\n", props[iDevice].name);
	printf(" - Compute capability: %d.%d\n", props[iDevice].major, props[iDevice].minor);
	printf(" - Clock rate: %d Hz\n", props[iDevice].clockRate);

	printf(" - Device copy overlap: ");
	if (props[iDevice].deviceOverlap)
		printf("Enabled!\n");
	else
		printf("Disabled!\n");

	printf(" - Kernel execution timeout: ");
	if (props[iDevice].kernelExecTimeoutEnabled)
		printf("Enabled\n");
	else
		printf("Disabled!\n");

	printf(" - Total global mem: %ld bytes\n", props[iDevice].totalGlobalMem);
	printf(" - Total constant mem: %ld bytes\n", props[iDevice].totalConstMem);
	printf(" - Max mem pitch: %ld\n", props[iDevice].memPitch);
	printf(" - Texture alignment: %ld\n", props[iDevice].textureAlignment);
	printf(" - Multiprocessor count: %d\n", props[iDevice].multiProcessorCount);
	printf(" - Shared memory per MP: %d bytes\n", props[iDevice].sharedMemPerBlock);
	printf(" - Registers per MP: %d\n", props[iDevice].regsPerBlock);
	printf(" - Threads in warp: %d\n", props[iDevice].warpSize);
	printf(" - Max threads per block: %d\n", props[iDevice].maxThreadsPerBlock);
	printf(" - Max thread dimensions: (%d, %d, %d)\n",
		props[iDevice].maxThreadsDim[0], 
		props[iDevice].maxThreadsDim[1], 
		props[iDevice].maxThreadsDim[2]
		);
	printf(" - Max grid dimensions: (%d, %d, %d)\n",
		props[iDevice].maxGridSize[0], 
		props[iDevice].maxGridSize[1], 
		props[iDevice].maxGridSize[2]
		);


	printf("\n");
	return;
}

void cudaTools::checkCudaErr(const hipError_t& err, const char* msgErr)
{
	if (err != hipSuccess)
	{
		printf("There was some CUDA error: %s, %s\n",
			msgErr, hipGetErrorString(err));
		throw "CudaError";
	}
	return;
}