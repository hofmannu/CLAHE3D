#include "cudaTools.cuh"

void cudaTools::checkCudaErr(hipError_t err, const char* msgErr)
{
	if (err != hipSuccess)
	{
		printf("There was some CUDA error: %s, %s\n",
			msgErr, hipGetErrorString(err));
		throw "CudaError";
	}
	return;
}