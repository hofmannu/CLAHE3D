#include "cudaTools.cuh"

void cudaTools::checkCudaErr(hipError_t err, const char* msgErr)
{
	if (err != hipSuccess)
	{
		printf("There was some CUDA error appearing along my way: %s\n",
			hipGetErrorString(err));
		throw "CudaError";
	}
	return;
}